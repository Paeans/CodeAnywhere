#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>

using namespace std;

__global__ void d_matrixm(double* matrix, double* result);

int main(int argc, char* argv[]){

  int matrix_size = atoi(argv[1]);
  int mem_size = sizeof(double) * matrix_size * matrix_size;
  double* matrix = (double*)malloc(mem_size);
  double* result = (double*)malloc(mem_size);
  
  ifstream infile;
  infile.open("matrix.txt");
  double d = 0;
  for(int i=0; i < matrix_size * matrix_size; i++){
      infile >> d;
      matrix[i] = result[i] = d;
      //cout << d << " ";
    }
  }
  infile.close();
  
  double *d_matrix_1, *d_matrix_2, *d_result;
  hipMalloc(&d_matrix_1, mem_size);
  hipMalloc(&d_matrix_2, mem_size);
  hipMalloc(&d_result, mem_size);
  
  int multime = 2;
  for(int i=0; i<multime; i++){
    hipMemcpy(d_matrix_1, matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_2, result, hipMemcpyHostToDevice);
    d_matrixm<<< matrix_size, matrix_size >>>(d_matrix_1, d_matrix_2, d_result);
    hipMemcpy(result, d_result, hipMemcpyDeviceToHost);
  }
  
  for(int i=0; i < matrix_size * matrix_size; i++){
    cout << result[i];
  }
  cout << endl;
  return 0;
}

__global__ void d_matrixm(double* matrix_1, double matrix_2, double* result){

  int size = blockDim.x;
  for(int i = 0; i<size; i++){
    result[ blockIdx.x * size + threadIdx ] += 
        matrix_1[ blockIdx.x * size + i ] * matrix_2[ threadIdx.x + i * size ];
  }
}

