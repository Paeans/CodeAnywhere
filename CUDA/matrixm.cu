#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void d_matrixm(double* matrix_1, double* matrix_2, double* result);

int main(int argc, char* argv[]){

	int matrix_size = atoi(argv[1]);
	int mem_size = sizeof(double) * matrix_size * matrix_size;
	double* matrix = (double*)malloc(mem_size);
	double* result = (double*)malloc(mem_size);

	ifstream infile;
	infile.open("matrix.txt");
	double d = 0;
	for (int i = 0; i < matrix_size * matrix_size; i++){
		infile >> d;
		matrix[i] = result[i] = d;
		cout << d << " ";	
	}
	cout << endl;
	infile.close();

	double *d_matrix_1, *d_matrix_2, *d_result;
	hipMalloc(&d_matrix_1, mem_size);
	hipMalloc(&d_matrix_2, mem_size);
	hipMalloc(&d_result, mem_size);

	int multime = 2;
	for (int i = 0; i<multime; i++){
		hipMemcpy(d_matrix_1, result, mem_size, hipMemcpyHostToDevice);
		hipMemcpy(d_matrix_2, matrix, mem_size, hipMemcpyHostToDevice);
		d_matrixm << < matrix_size, matrix_size >> >(d_matrix_1, d_matrix_2, d_result);
		hipMemcpy(result, d_result, mem_size, hipMemcpyDeviceToHost);
	}

	for (int i = 0; i < matrix_size * matrix_size; i++){
		cout << result[i] << " ";
	}
	cout << endl;
	return 0;
}

__global__ void d_matrixm(double* matrix_1, double* matrix_2, double* result){

	int size = blockDim.x;
	result[blockIdx.x * size + threadIdx.x] = 0;
	for (int i = 0; i < size; i++){
		result[blockIdx.x * size + threadIdx.x] +=
			matrix_1[blockIdx.x * size + i] * matrix_2[threadIdx.x + i * size];
	}	
}
