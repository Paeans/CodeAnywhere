
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

using namespace std;

#define CUDA true

__global__ void multiadd(int N, float a, float* x, float* y);

int main(int argc, char* argv[]){

	cout << "This is test CUDA in visual studio!" << endl;

	int N = 1 << 20;
	int size = sizeof(float) * N;
	float* x = (float*)malloc(size);
	float* y = (float*)malloc(size);
	float *dx, *dy;
	hipMalloc(&dx, size);
	hipMalloc(&dy, size);

	for (int i = 0; i < N; i++){
		x[i] = 1.0;
		y[i] = 2.0;
	}
	if (CUDA){
		hipMemcpy(dx, x, size, hipMemcpyHostToDevice);
		hipMemcpy(dy, y, size, hipMemcpyHostToDevice);

		multiadd <<< (N + 255) / 256, 256 >>>(N, 3, dx, dy);

		hipMemcpy(y, dy, size, hipMemcpyDeviceToHost);
	}
	else{
		for (int i = 0; i < N; i++){
			y[i] = 3 * x[i] + y[i];
		}
	}

	int counter = 0;
	for (int i = 0; i < N; i++){
		if (y[i] != 5.0) counter++;
	}
	cout << "Error number is: " << counter << endl;

	return 0;
}

__global__ void multiadd(int N, float a, float* x, float* y){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) y[i] = a * x[i] + y[i];
}

